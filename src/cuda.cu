#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_cooperative_groups.h>


//CHECK FUNCTION
__host__ void checkStatus(hipError_t cudaStatus, const char err[])
{
    if(cudaStatus != hipSuccess)
    {
        printf("%s",err);
        exit(1);
    }
}

__device__ int checkCoservativeGroup(char seq1 , char seq2)
{
    const char* CONSERVATIVE_GROUP[] = {"NDEQ" , "NEQK" , "STA", "FYW",
        "MILV", "MILF" , "QHRK" , "NHQK" , "HY"};
	//strlen(CONSERVATIVE_GROUP) doesn't work, went for 9
    int j ,k;
	for(int i = 0 ; i < 9 ; i++){
        j=0;
        k=0;
        while(CONSERVATIVE_GROUP[i][j] != '\0')
        {
            if(CONSERVATIVE_GROUP[i][j] == seq1)
            {
                 while(CONSERVATIVE_GROUP[i][k] != '\0')
                {
                    if(CONSERVATIVE_GROUP[i][k] == seq2)
                    {
                        return 1;
                    }
                    k++;
                }
            }
            j++;
        }
	}
	return 0;
}




__device__ int checkSemiCoservativeGroup(char seq1 , char seq2)
{
    const char* SEMI_CONSERVATIVE_GROUPS[] = {"SAG" , "ATV", "CSA" , "SGND", "STPA",
        "STNK" , "NEQHRK" , "NDEQHK" , "SNDEQK" , "HFY" , "FVLIM"};
    int j,k;

	for(int i = 0 ; i < 11 ; i++){
        j=0;
        k=0;
        while(SEMI_CONSERVATIVE_GROUPS[i][j] != '\0')
        {
            if(SEMI_CONSERVATIVE_GROUPS[i][j] == seq1)
            {
                while(SEMI_CONSERVATIVE_GROUPS[i][k] != '\0')
                {
                    if(SEMI_CONSERVATIVE_GROUPS[i][k] == seq2)
                    {
                        return 1;
                    }
                    k++;
                }
            }
            j++;
        }
	}
	return 0;
}


__device__ void calcScore(float* weights, Mutant* currentMutant, char* seq1 , 
                char* seq2, int  sizeOfSeq2, int seq1Index)
{
	int spaces = 0;
    int points = 0;
    int colons = 0;
    int stars = 0;


	for(int seq2Index = 0 ; seq2Index < sizeOfSeq2 ; seq2Index++)
	{
		if(seq2Index == currentMutant->n || seq2Index == currentMutant->k){
			continue;
		}
		else if(seq1[seq1Index] == seq2[seq2Index])
		{
			stars++;
			seq1Index++;

		}
		else if(checkCoservativeGroup(seq1[seq1Index],seq2[seq2Index]))
		{
			colons++;
			seq1Index++;

		}
		else if(checkSemiCoservativeGroup(seq1[seq1Index],seq2[seq2Index]))
		{
			points++;
			seq1Index++;

		}
		else{
			spaces++;
			seq1Index++;
		}
	}

	currentMutant->score = (float)(weights[0]*stars) - (float)(weights[1]*colons) - (float)(weights[2]*points) - (float)(weights[3]*spaces);
}



__device__ void getMax(Mutant* mutants,Mutant* best , int numOfMutants){
	Mutant max = mutants[0];
	for (int i = 1; i < numOfMutants; i++) {
		if(mutants[i].score>max.score){
			max = mutants[i];
		}
	}
	best->score = max.score;
	best->k = max.k;
	best->n = max.n;
}



__global__ void cudaCalculateBestScore (float* weights ,char* seq1, char* seq2, Mutant* cudaMutants , int sizeOfSeq2 
                                ,int seq1Length , Mutant* bestMutant , int offset)
{

    //each i represents a mutant
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int sizeOfMutants = (sizeOfSeq2*(sizeOfSeq2-1))/2;
    
    //each mutant's score will be calcualted if he's lower than max num of mutants
    if (i < sizeOfMutants)
        calcScore(weights, &cudaMutants[i] ,seq1 , seq2, sizeOfSeq2, offset);

    //__syncthreads();

    //getting random mutant to calculate the bestMutant out of all the mutants:
    if(i==0)
        getMax(cudaMutants , bestMutant , sizeOfMutants);

    //__syncthreads();
}  



void cudaKernal(Mutant* allMutants , char* seq1, char* seq2 , int sizeOfSeq2 ,int seq1Length, float*  weights
                     ,Mutant* currentMutant)

{
    //variables for cuda
    hipError_t cudaStatus;
    char* cudaSeq1;
    char* cudaSeq2;
    Mutant* cudaMutants = NULL;
    float* cudaWeights;
    Mutant* bestMutant = NULL;

    //num of mutants
    int numOfMutants = (sizeOfSeq2*(sizeOfSeq2-1))/2;

    //sizes of each cuda variable
    size_t sizeSeq1 = (seq1Length) * sizeof(char);
    size_t sizeSeq2 = (sizeOfSeq2) * sizeof(char);
    size_t sizeWeights = 4 * sizeof(float);
    int sizeMutants = numOfMutants * sizeof(Mutant);

    //num of blocks and num of threads per block
    int threadsPerBlock = MAX_THREADS;
    int blocksPerGrid = (numOfMutants + threadsPerBlock - 1) / threadsPerBlock;

    //copying and allocating seq2 to cuda:
    cudaStatus = hipMalloc((void**)&cudaSeq2,sizeSeq2);
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(cudaSeq2,seq2,sizeSeq2,hipMemcpyHostToDevice);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");
    
    //copying  and allocating seq1 to cuda:
	cudaStatus = hipMalloc((void**)&cudaSeq1,sizeSeq1);
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(cudaSeq1,seq1,sizeSeq1,hipMemcpyHostToDevice);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //copying  and allocating array of all mutants (that each of them contains n,k) to cuda:
	cudaStatus = hipMalloc((void**)&cudaMutants,sizeMutants);
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(cudaMutants,allMutants,sizeMutants,hipMemcpyHostToDevice);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //copying weights to cuda:
	cudaStatus = hipMalloc((void**)&cudaWeights,sizeWeights);
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(cudaWeights,weights,sizeWeights,hipMemcpyHostToDevice);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //copying and allocating bestmutant to cuda:
	cudaStatus = hipMalloc((void**)&bestMutant,sizeof(Mutant));
    checkStatus(cudaStatus,"Cuda Malloc Failed!");
    cudaStatus = hipMemcpy(bestMutant,currentMutant,sizeof(Mutant),hipMemcpyHostToDevice);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");

    //calling global func to find best mutant
    cudaCalculateBestScore<<<blocksPerGrid,threadsPerBlock>>>(cudaWeights ,cudaSeq1,cudaSeq2,cudaMutants ,sizeOfSeq2 
                                    ,seq1Length , bestMutant , currentMutant->offset);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, "Cuda Failed!");

    //copying best mutant from cuda to current mutant of given offset in openmp
    cudaStatus = hipMemcpy(currentMutant,bestMutant,sizeof(Mutant),hipMemcpyDeviceToHost);
    checkStatus(cudaStatus, "Cuda MEMCPY failed!");


    //Free all allocations from cuda
    cudaStatus = hipFree(cudaSeq2);
    checkStatus(cudaStatus,"Cuda Free Failed!");
    cudaStatus = hipFree(cudaSeq1);
    checkStatus(cudaStatus,"Cuda Free Failed!");
    cudaStatus = hipFree(cudaWeights);
    checkStatus(cudaStatus,"Cuda Free Failed!");
    cudaStatus = hipFree(cudaMutants);
    checkStatus(cudaStatus,"Cuda Free Failed!");
    cudaStatus = hipFree(bestMutant);
    checkStatus(cudaStatus,"Cuda Free Failed!");
}


